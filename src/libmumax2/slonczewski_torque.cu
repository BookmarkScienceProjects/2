#include "hip/hip_runtime.h"
#include "slonczewski_torque.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include <hip/hip_runtime.h>

#ifdef __cplusplus
extern "C" {
#endif
  // ========================================

  __global__ void slonczewski_deltaMKern(float* sttx, float* stty, float* sttz, 
					 float* mx, float* my, float* mz, 
					 float* px, float* py, float* pz,
					 float* alpha, float* Msat,
					 float gamma, float aj, float bj, float Pol, 
					 float *curr, 
					 int NPart)
  {
    
    int I = threadindex;
	if (I < NPart){ // Thread configurations are usually too large...

      //float Ms = Msat[I];
    
    //if (Ms > 0.0) { // do bother if there's nothing here, branching makes cuda code slower.
      float m_x = mx[I];
      float m_y = my[I];
      float m_z = mz[I];
      float p_x = px[I];
      float p_y = py[I];
      float p_z = pz[I];

      float pxm_x = -p_y * m_z + m_y * p_z;
      float pxm_y =  p_x * m_z - m_x * p_z;
      float pxm_z = -p_x * m_y + m_x * p_y;
      
      float mxpxm_x = -pxm_y * m_z + m_y * pxm_z;
      float mxpxm_y =  pxm_x * m_z - m_x * pxm_z;
      float mxpxm_z = -pxm_x * m_y + m_x * pxm_y;

      sttx[I] = mxpxm_x;//0.0*mxpxm_x;
      stty[I] = mxpxm_y;//0.0*mxpxm_y;
      sttz[I] = mxpxm_z;//0.0*mxpxm_z;
      
    } 
        
  }

  #define BLOCKSIZE 16
  
  void slonczewski_async(float** sttx, float** stty, float** sttz, 
			 float** mx, float** my, float** mz, 
			 float** px, float** py, float** pz,
			 float** alpha, float** Msat,
			 float gamma, float aj, float bj, float Pol,
			 float **curr, 
			 int NPart, 
			 hipStream_t* stream)
  {

    // 1D configuration
    dim3 gridSize, blockSize;
    make1dconf(NPart, &gridSize, &blockSize);

    int nDev = nDevice();
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));
	    slonczewski_deltaMKern<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (sttx[dev], stty[dev], sttz[dev],  
										       mx[dev], my[dev], mz[dev],  
										       px[dev], py[dev], pz[dev],
										       alpha[dev], Msat[dev], gamma, aj, bj, Pol, curr[dev], 
										       NPart);
    } // end dev < nDev loop
										  
										  
  }

  // ========================================

#ifdef __cplusplus
}
#endif
