#include "hip/hip_runtime.h"
#include "normalize.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void normalizeKern(float* mx, float* my, float* mz, 
						   float* norm_map, int Npart) {
	int i = threadindex;
	if (i < Npart) {

		// reconstruct inverse norm from map
		float invnorm;
		if(norm_map == NULL){
			invnorm = 1.0f;
		}else{
			invnorm = norm_map[i];
			if(invnorm != 0.0f){
				invnorm = 1.0f/invnorm;
			}
		}

    	float Mx = mx[i];
    	float My = my[i];
    	float Mz = mz[i];
    
		invnorm = invnorm * (1.0f/sqrtf(Mx*Mx + My*My + Mz*Mz));	
		mx[i] = Mx * invnorm;
		my[i] = My * invnorm;
		mz[i] = Mz * invnorm;
	}
}


void normalizeAsync(float** mx, float** my, float** mz, float** norm_map, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int i = 0; i < nDevice(); i++) {
		assert(mx[i] != NULL);
		assert(my[i] != NULL);
		assert(mz[i] != NULL);
		// normMap may be null
		gpu_safe(hipSetDevice(deviceId(i)));
		normalizeKern <<<gridSize, blockSize, 0, hipStream_t(stream[i])>>> (mx[i],my[i],mz[i], norm_map[i], Npart);
	}
}

#ifdef __cplusplus
}
#endif
