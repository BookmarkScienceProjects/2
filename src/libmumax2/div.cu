#include "hip/hip_runtime.h"
#include "div.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void divKern(float* dst, float* a, float* b, int Npart) {
	int i = threadindex;
	if (i < Npart) {

        float bb = (b == NULL) ? 1.0f : b[i];
		dst[i] = (bb == 0.0f)? 0.0f : a[i] / bb;
	}
}


__export__ void divAsync(float** dst, float** a, float** b, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		divKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], a[dev], b[dev], Npart);
	}
}

#ifdef __cplusplus
}
#endif

