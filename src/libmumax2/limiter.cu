#include "hip/hip_runtime.h"
#include "normalize.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void limiterKern(float* __restrict__ Mx, float* __restrict__ My, float* __restrict__ Mz, 
                            float* __restrict__ limitMask,
                            float msatMul,
			                float limitMul, 
			                int Npart) {
	int i = threadindex;
	
	if (i < Npart) {

		float3 M = make_float3(Mx[i], My[i], Mz[i]);

		float nMn = len(M);

		float limit = (limitMask != NULL) ? limitMask[i] * limitMul : limitMul;

		if (nMn == 0.0f || limit == 0.0f) {
		    Mx[i] = 0.0f;
		    My[i] = 0.0f;
		    Mz[i] = 0.0f;
		    return;
		}
	    		
	    float Ms = msatMul * nMn;
	       			
		float ratio = limit / Ms;        

		float norm = (ratio < 1.0f) ? ratio : 1.0f;
		
		Mx[i] = M.x * norm;
		My[i] = M.y * norm;
		Mz[i] = M.z * norm;	
	}
}


__export__ void limiterAsync(float** Mx, float** My, float** Mz,
                             float** limitMask,
                             float msatMul,
                             float limitMul,
                             hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		limiterKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (Mx[dev],My[dev],Mz[dev],
		                                                                     limitMask[dev],
		                                                                     msatMul,
		                                                                     limitMul,
		                                                                     Npart);
	}
}

#ifdef __cplusplus
}
#endif
