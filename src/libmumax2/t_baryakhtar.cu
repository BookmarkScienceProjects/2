#include "hip/hip_runtime.h"
#include "t_baryakhtar.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif
   
 __global__ void tbaryakhtar_delta2HKernMGPU(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                     float* __restrict__ l,
					 float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
					 float* __restrict__ hx, float* __restrict__ hy, float* __restrict__ hz,
					 float* __restrict__ lhx, float* __restrict__ lhy, float* __restrict__ lhz,
					 float* __restrict__ rhx, float* __restrict__ rhy, float* __restrict__ rhz,	 
					 float* __restrict__ msat,
					 float* __restrict__ AexMsk,
					 float* __restrict__ alphaMsk,
					 const float alphaMul,
					 const float pred,
					 const float pre,
					 const float pret,
					 const int4 size,		
					 const float3 mstep,
					 const int3 pbc,
					 const int i)
  {	
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;			
	int x0 = i * size.w + j * size.z + k;
		    
	float m_sat = (msat != NULL) ? msat[x0] : 1.0f;
		
	if (m_sat == 0.0f){
	    tx[x0] = 0.0f;
	    ty[x0] = 0.0f;
	    tz[x0] = 0.0f;
	    
	    l[x0] = 0.0f;
	    return;
	}
	
    if (j < size.y && k < size.z){ // 3D now:)
        
	    
        float A = (AexMsk != NULL) ? AexMsk[x0] : 1.0f;
        float alpha = (alphaMsk != NULL) ? alphaMsk[x0] * alphaMul : alphaMul;
        
        /*if (x0 == 100) {
	        printf("msat: %e  ", m_sat);
	        printf("pre: %e  ", pre);
	        printf("A: %e  ", A);
	        printf("alpha: %e  ", alpha);
	    }*/
	    
	    
        m_sat = pred * A / (m_sat * m_sat * (1.0f + alpha * alpha)); // pred is with Gilbert's gamma
        
        float prel =  pre * A / (m_sat * m_sat); // pre is without Gilbert gamma
        
        float3 m = make_float3(mx[x0], my[x0], mz[x0]);		
        
        // Longitudinal part
        
        float3 h = make_float3(hx[x0], hy[x0], hz[x0]);
        
        float lr = pret * dotf(h, m); // lambda * (H, m)   
        
        // Transverse part    
         
        
        // Second-order derivative 5-points stencil

        int xb2 = i - 2;
        int xb1 = i - 1;
        int xf1 = i + 1;
        int xf2 = i + 2;

        int yb2 = j - 2;
        int yb1 = j - 1;
        int yf1 = j + 1;
        int yf2 = j + 2; 

        int zb2 = k - 2;
        int zb1 = k - 1;
        int zf1 = k + 1;
        int zf2 = k + 2;

        int4 yi = make_int4(yb2, yb1, yf1, yf2);		  

        xb2 = (pbc.x == 0 && xb2 < 0)? i : xb2; // backward coordinates are negative
        xb1 = (pbc.x == 0 && xb1 < 0)? i : xb1;
        xf1 = (pbc.x == 0 && xf1 >= size.x)? i : xf1;
        xf2 = (pbc.x == 0 && xf2 >= size.x)? i : xf2;
          
        yb2 = (lhx == NULL && yb2 < 0)? j : yb2;
        yb1 = (lhx == NULL && yb1 < 0)? j : yb1;
        yf1 = (rhx == NULL && yf1 >= size.y)? j : yf1;
        yf2 = (rhx == NULL && yf2 >= size.y)? j : yf2;
          
        zb2 = (pbc.z == 0 && zb2 < 0)? k : zb2;
        zb1 = (pbc.z == 0 && zb1 < 0)? k : zb1;
        zf1 = (pbc.z == 0 && zf1 >= size.z)? k : zf1;
        zf2 = (pbc.z == 0 && zf2 >= size.z)? k : zf2;
          
        xb2 = (xb2 >= 0)? xb2 : size.x + xb2;
        xb1 = (xb1 >= 0)? xb1 : size.x + xb1;
        xf1 = (xf1 < size.x)? xf1 : xf1 - size.x;
        xf2 = (xf2 < size.x)? xf2 : xf2 - size.x;

        yb2 = (yb2 >= 0)? yb2 : size.y + yb2;
        yb1 = (yb1 >= 0)? yb1 : size.y + yb1;
        yf1 = (yf1 < size.y)? yf1 : yf1 - size.y;
        yf2 = (yf2 < size.y)? yf2 : yf2 - size.y;

        zb2 = (zb2 >= 0)? zb2 : size.z + zb2;
        zb1 = (zb1 >= 0)? zb1 : size.z + zb1;
        zf1 = (zf1 < size.z)? zf1 : zf1 - size.z;
        zf2 = (zf2 < size.z)? zf2 : zf2 - size.z;
          
        int comm = j * size.z + k;	   
        int4 xn = make_int4(xb2 * size.w + comm, 
				          xb1 * size.w + comm, 
				          xf1 * size.w + comm, 
				          xf2 * size.w + comm); 
				         

        comm = i * size.w + k; 
        int4 yn = make_int4(yb2 * size.z + comm, 
				          yb1 * size.z + comm, 
				          yf1 * size.z + comm, 
				          yf2 * size.z + comm);


        comm = i * size.w + j * size.z;
        int4 zn = make_int4(zb2 + comm, 
				          zb1 + comm, 
				          zf1 + comm, 
				          zf2 + comm);


        // Let's use 5-point stencil to avoid problems at the boundaries
        // CUDA does not have vec3 operators like GLSL has, except of .xxx, 
        // Perhaps for performance need to take into account special cases where j || to x, y or z  

        float4 HH;

        HH.x = (yi.x >= 0 || lhx == NULL) ? hx[yn.x] : lhx[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hx[yn.y] : lhx[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hx[yn.z] : rhx[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hx[yn.w] : rhx[yn.w];
          	    
        float3 dhxdr2 = 	make_float3(mstep.x * (-hx[xn.x] + 16.0f * hx[xn.y] - 30.0f * hx[x0] + 16.0f * hx[xn.z] - hx[xn.w]),
							        mstep.y * (-HH.x + 16.0f * HH.y - 30.0f * hx[x0] + 16.0f * HH.z - HH.w),
							        mstep.z * (-hx[zn.x] + 16.0f * hx[zn.y] - 30.0f * hx[x0] + 16.0f * hx[zn.z] - hx[zn.w]));
							
        HH.x = (yi.x >= 0 || lhx == NULL) ? hy[yn.x] : lhy[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hy[yn.y] : lhy[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hy[yn.z] : rhy[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hy[yn.w] : rhy[yn.w];
						              
        float3 dhydr2 = 	make_float3(mstep.x * (-hy[xn.x] + 16.0f * hy[xn.y] - 30.0f * hy[x0] + 16.0f * hy[xn.z] - hy[xn.w]),
						            mstep.y * (-HH.x + 16.0f * HH.y - 30.0f * hy[x0] + 16.0f * HH.z - HH.w),
							        mstep.z * (-hy[zn.x] + 16.0f * hy[zn.y] - 30.0f * hy[x0] + 16.0f * hy[zn.z] - hy[zn.w]));
							
        HH.x = (yi.x >= 0 || lhx == NULL) ? hz[yn.x] : lhz[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hz[yn.y] : lhz[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hz[yn.z] : rhz[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hz[yn.w] : rhz[yn.w]; 								
							
								
        float3 dhzdr2 = 	make_float3(mstep.x * (-hz[xn.x] + 16.0f * hz[xn.y] - 30.0f * hz[x0] + 16.0f * hz[xn.z] - hz[xn.w]),
							        mstep.y * (-HH.x + 16.0f * HH.y - 30.0f * hz[x0] + 16.0f * HH.z - HH.w),
						            mstep.z * (-hz[zn.x] + 16.0f * hz[zn.y] - 30.0f * hz[x0] + 16.0f * hz[zn.z] - hz[zn.w])); 


	          
        float3 ddh = make_float3(dhxdr2.x + dhxdr2.y + dhxdr2.z, dhydr2.x + dhydr2.y + dhydr2.z, dhzdr2.x + dhzdr2.y + dhzdr2.z);
        
		// Longitudinal part			
	    float le = prel * dotf(m, ddh); // Lambda_e * (m, laplace(h) 
	    l[x0] = lr - le; 
	    //*****************    	  
	    
        float3 ddhxm = crossf(m, ddh); // no minus in it, but it was an interesting behaviour when damping is pumping

        float3 mxddhxm = crossf(m, ddhxm); // with plus from [ddh x m]
       
         
        tx[x0] = m_sat * mxddhxm.x;
        ty[x0] = m_sat * mxddhxm.y;
        tz[x0] = m_sat * mxddhxm.z;  
        

    } 
  }

  
#define BLOCKSIZE 16


  
__export__  void tbaryakhtar_async(float** tx, float**  ty, float**  tz, 
             float** l,
			 float**  mx, float**  my, float**  mz, 
			 float**  hx, float**  hy, float**  hz,
			 float**  msat,
			 float**  AexMsk,
			 float**  alphaMsk,
			 const float alphaMul,
			 const float pred,
			 const float pre,
			 const float pret,
			 const int sx, const int sy, const int sz,
			 const float csx, const float csy, const float csz,
			 const int pbc_x, const int pbc_y, const int pbc_z, 
			 hipStream_t* stream)
  {

	// 3D :)
	
	dim3 gridSize(divUp(sy, BLOCKSIZE), divUp(sz, BLOCKSIZE));
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
		
	// FUCKING THREADS PER BLOCK LIMITATION
	check3dconf(gridSize, blockSize);
		
	float i12csx = 1.0f / (12.0f * csx * csx);
	float i12csy = 1.0f / (12.0f * csy * csy);
	float i12csz = 1.0f / (12.0f * csz * csz);
	
	int syz = sy * sz;
	
		
	float3 mstep = make_float3(i12csx, i12csy, i12csz);	
	int4 size = make_int4(sx, sy, sz, syz);
	int3 pbc = make_int3(pbc_x, pbc_y, pbc_z);
	
    int nDev = nDevice();
		
	/*hipEvent_t start,stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);*/
		
	
	
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));	 
	  		
		// calculate dev neighbours
		
		int ld = Mod(dev - 1, nDev);
		int rd = Mod(dev + 1, nDev);
				
		float* lhx = hx[ld]; 
		float* lhy = hy[ld];
		float* lhz = hz[ld];

		float* rhx = hx[rd]; 
		float* rhy = hy[rd];
		float* rhz = hz[rd];
		
		if(pbc_y == 0){             
			if(dev == 0){
				lhx = NULL;
				lhy = NULL;
				lhz = NULL;			
			}
			if(dev == nDev-1){
				rhx = NULL;
				rhy = NULL;
				rhz = NULL;
			}
		}
		
		// printf("Devices are: %d\t%d\t%d\n", ld, dev, rd);
		
		for (int i = 0; i < sx; i++) {
												   
			tbaryakhtar_delta2HKernMGPU<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (tx[dev], ty[dev], tz[dev],  
			                                       l[dev],
												   mx[dev], my[dev], mz[dev],												   
												   hx[dev], hy[dev], hz[dev],
												   lhx, lhy, lhz,
												   rhx, rhy, rhz,
												   msat[dev],
												   AexMsk[dev],							 
												   alphaMsk[dev],
												   alphaMul,
												   pred,
												   pre,
												   pret,
												   size,
												   mstep,
												   pbc,
												   i);
		}

    } // end dev < nDev loop
	
	
	/*hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("T-Baryakhtar kernel requires: %f ms\n",time);*/
	
  }
  
  // ========================================

#ifdef __cplusplus
}
#endif
