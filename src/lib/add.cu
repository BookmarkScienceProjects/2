#include "hip/hip_runtime.h"

#include "add.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void addKern(float* dst, float* a, float* b, int Npart) {
	int i = threadindex;
	if (i < Npart) {
		dst[i] = a[i] + b[i];
	}
}


void addAsync(float** dst, float** a, float** b, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		addKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], a[dev], b[dev], Npart);
	}
}



///@internal
__global__ void maddKern(float* dst, float* a, float* b, float mulB, int Npart) {
	int i = threadindex;
	float bMask;
	if (b == NULL){
		bMask = 1.0f;
	}else{
		bMask = b[i];
	}
	if (i < Npart) {
		dst[i] = a[i] + mulB * bMask;
	}
}


void maddAsync(float** dst, float** a, float** b, float mulB, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		maddKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], a[dev], b[dev], mulB, Npart);
	}
}

#ifdef __cplusplus
}
#endif
