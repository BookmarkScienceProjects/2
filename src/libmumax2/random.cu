#include "hip/hip_runtime.h"
#include "random.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#define SEED 123
#define OFFSET 0


///@internal
__global__ void setUpRandomRegionKern(hiprandState* state)
{
    int i = threadindex;
    hiprand_init(SEED, i, OFFSET, &state[i]);
}

///@internal
__global__ void initScalarQuantRandomUniformRegionKern(float* S,
        float* regions,
        bool* regionsToProceed,
        int regionNum,
        int Npart,
        hiprandState* globalState,
        float max, float min)
{
    int i = threadindex;
    if (i < Npart)
    {
        int regionIndex = __float2int_rn(regions[i]);
        if (regionIndex < regionNum && regionIndex > 0 && regionsToProceed[regionIndex] == true)
        {
            hiprandState localState = globalState[i];
            S[i] = min + (max - min) * hiprand_uniform(&localState);
            globalState[i] = localState;
        }
    }
}


__export__ void initScalarQuantRandomUniformRegionAsync(float** S,
        float** regions,
        bool* host_regionsToProceed,
        int regionNum,
        hipStream_t* stream,
        int Npart,
        float max, float min)
{
    assert(max != min);
    if (max < min)
    {
        float tmp = min;
        min = max;
        max = tmp;
    }
    hiprandState* devState;
    bool* dev_regionsToProceed;
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    for (int dev = 0; dev < nDevice(); dev++)
    {
        assert(S[dev] != NULL);
        assert(regions[dev] != NULL);
        assert(host_regionsToProceed != NULL);
        gpu_safe(hipSetDevice(deviceId(dev)));
        gpu_safe( hipMalloc( (void**)&dev_regionsToProceed, regionNum * sizeof(bool)));
        gpu_safe( hipMemcpy(dev_regionsToProceed, host_regionsToProceed, regionNum * sizeof(bool), hipMemcpyHostToDevice));
        gpu_safe( hipMalloc( (void**)&devState, Npart * sizeof(hiprandState)));
        setUpRandomRegionKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (devState);
        initScalarQuantRandomUniformRegionKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (S[dev], regions[dev], dev_regionsToProceed, regionNum, Npart, devState, max, min);
        hipFree(dev_regionsToProceed);
        hipFree(devState);
    }
}

///@internal
__global__ void initVectorQuantRandomUniformRegionKern(float* Sx, float* Sy, float* Sz,
        float* regions,
        bool* regionsToProceed,
        int regionNum,
        int Npart,
        hiprandState* globalState)
{
    int i = threadindex;
    if (i < Npart)
    {
        int regionIndex = __float2int_rn(regions[i]);
        if (regionIndex < regionNum && regionIndex > 0 && regionsToProceed[regionIndex] == true)
        {
            hiprandState localState = globalState[i];
            Sx[i] = 2.0f * hiprand_uniform(&localState) - 1.0f;
            Sy[i] = 2.0f * hiprand_uniform(&localState) - 1.0f;
            Sz[i] = 2.0f * hiprand_uniform(&localState) - 1.0f;
            float norm = sqrt(Sx[i] * Sx[i] + Sy[i] * Sy[i] + Sz[i] * Sz[i]);
            Sx[i] /= norm;
            Sy[i] /= norm;
            Sz[i] /= norm;
            globalState[i] = localState;
        }
    }
}


__export__ void initVectorQuantRandomUniformRegionAsync(float** Sx, float** Sy, float** Sz,
        float** regions,
        bool* host_regionsToProceed,
        int regionNum,
        hipStream_t* stream,
        int Npart)
{
    hiprandState* devState;
    bool* dev_regionsToProceed;
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    for (int dev = 0; dev < nDevice(); dev++)
    {
        assert(Sx[dev] != NULL);
        assert(Sy[dev] != NULL);
        assert(Sz[dev] != NULL);
        assert(regions[dev] != NULL);
        assert(host_regionsToProceed != NULL);
        gpu_safe(hipSetDevice(deviceId(dev)));
        gpu_safe( hipMalloc( (void**)&dev_regionsToProceed, regionNum * sizeof(bool)));
        gpu_safe( hipMemcpy(dev_regionsToProceed, host_regionsToProceed, regionNum * sizeof(bool), hipMemcpyHostToDevice));
        gpu_safe( hipMalloc( (void**)&devState, Npart * sizeof(hiprandState)));
        setUpRandomRegionKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (devState);
        initVectorQuantRandomUniformRegionKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (Sx[dev], Sy[dev], Sz[dev], regions[dev], dev_regionsToProceed, regionNum, Npart, devState);
        hipFree(dev_regionsToProceed);
        hipFree(devState);
    }
}

#ifdef __cplusplus
}
#endif
