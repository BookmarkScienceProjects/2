#include "hip/hip_runtime.h"
//  This file is part of MuMax, a high-performance micromagnetic simulator.
//  Copyright 2011  Arne Vansteenkiste and Ben Van de Wiele.
//  Use of this source code is governed by the GNU General Public License version 3
//  (as published by the Free Software Foundation) that can be found in the license.txt file.
//  Note that you are welcome to modify this code under the condition that you do not remove any 
//  copyright notices and prominently state that you modified it, giving a relevant date.

/// This file implements GPU memory operations

#include "macros.h"

#ifdef __cplusplus
extern "C" {
#endif


/// Sets the first N elements of array to value.
__global__ void gpuMemSet(float value, float* array, int N){
	int i = threadindex;
	if(i < N){
		array[i] = value;
	}
}


#ifdef __cplusplus
}
#endif
