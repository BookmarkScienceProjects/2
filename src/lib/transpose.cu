#include "hip/hip_runtime.h"
/**
 * @author Arne Vansteenkiste
 */
#include "transpose.h"

#include "gpu_safe.h"
#include "gpu_conf.h"
#include "multigpu.h"
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif

typedef struct{
  float real;
  float imag;
}complex;

/// The size of matrix blocks to be loaded into shared memory.
#define BLOCKSIZE 16


/// @internal Single-GPU complex matrix transpose
__global__ void transposeComplexYZKernel(complex* output, complex* input, int N1, int N2, int N)
{
  __shared__ complex block[BLOCKSIZE][BLOCKSIZE+1];

  for (int x=0; x<N; x++){
    // index of the block inside the blockmatrix
    int BI = blockIdx.x;
    int BJ = blockIdx.y;

    // "minor" indices inside the tile
    int i = threadIdx.x;
    int j = threadIdx.y;

    {
      // "major" indices inside the entire matrix
      int I = BI * BLOCKSIZE + i;
      int J = BJ * BLOCKSIZE + j;

      if((I < N1) && (J < N2)){
        block[j][i] = input[x*N1*N2 + J * N1 + I];
      }
    }
    __syncthreads();

    {
      // Major indices with transposed blocks but not transposed minor indices
      int It = BJ * BLOCKSIZE + i;
      int Jt = BI * BLOCKSIZE + j;

      if((It < N2) && (Jt < N1)){
        output[x*N1*N2 + Jt * N2 + It] = block[i][j];
      }
    }
    __syncthreads();
  }
  
  return;
}


void transposeComplexYZAsyncPart(float** output, float** input, int N0, int N1, int N2, hipStream_t* stream){
    N2 /= 2; // number of complex
    dim3 gridsize((N2-1) / BLOCKSIZE + 1, (N1-1) / BLOCKSIZE + 1, 1); // integer division rounded UP. Yes it has to be N2, N1
    dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
  	transposeComplexYZKernel<<<gridsize, blocksize, 0, stream[dev]>>>((complex*)output[dev], (complex*)input[dev], N2, N1, N0);
	}
}


#ifdef __cplusplus
}
#endif

