#include "hip/hip_runtime.h"
#include "t_baryakhtar.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif
   
 __global__ void tbaryakhtar_delta2HKernMGPU(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                     float* __restrict__ l,
					 float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
					 float* __restrict__ hx, float* __restrict__ hy, float* __restrict__ hz,
					 float* __restrict__ lhx, float* __restrict__ lhy, float* __restrict__ lhz,
					 float* __restrict__ rhx, float* __restrict__ rhy, float* __restrict__ rhz,	 
					 float* __restrict__ msat,
					 const float msatMul,
					 const float lambda,
					 const float lambda_e,
					 const int4 size,		
					 const float3 mstep,
					 const int3 pbc,
					 const int i)
  {	
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;			
	int x0 = i * size.w + j * size.z + k;
		    
	float m_sat = (msat != NULL) ? msat[x0] * msatMul : msatMul;
	
	if (m_sat == 0.0f){
	    tx[x0] = 0.0f;
	    ty[x0] = 0.0f;
	    tz[x0] = 0.0f;
	    
	    l[x0] = 0.0f;
	    return;
	}
	
    if (j < size.y && k < size.z){ // 3D now:)
        
	    m_sat = 1.0f / m_sat;             
        
        float5 cfx = make_float5(-1.0f, +16.0f, -30.0f, +16.0f, -1.0f);
	    float5 cfy = make_float5(-1.0f, +16.0f, -30.0f, +16.0f, -1.0f);
	    float5 cfz = make_float5(-1.0f, +16.0f, -30.0f, +16.0f, -1.0f);
	    
	    float3 mmstep = mstep;
	    
	    if (pbc.x == 0 && i <= 1) {
            cfx.x = +0.0f;
            cfx.y = +0.0f;
            cfx.z = +1.0f;
            cfx.w = -2.0f;
            cfx.t = +1.0f;
            mmstep.x *= 12.0f;
        }
        
        if (pbc.x == 0 && i >= size.x - 2) {
            cfx.x = +1.0f;
            cfx.y = -2.0f;
            cfx.z = +1.0f;
            cfx.w = +0.0f;
            cfx.t = +0.0f;
            mmstep.x *= 12.0f;
        }  
              

        if (pbc.y == 0 && j <= 1) {
            cfy.x = +0.0f;
            cfy.y = +0.0f;
            cfy.z = +1.0f;
            cfy.w = -2.0f;
            cfy.t = +1.0f;
            mmstep.y *= 12.0f;
        }
        if (pbc.y == 0 && j >= size.y - 2) {
            cfy.x = +1.0f;
            cfy.y = -2.0f;
            cfy.z = +1.0f;
            cfy.w = +0.0f;
            cfy.t = +0.0f;
            mmstep.y *= 12.0f;
        }
        if (pbc.z == 0 && k <= 1) {
            cfz.x = +0.0f;
            cfz.y = +0.0f;
            cfz.z = +1.0f;
            cfz.w = -2.0f;
            cfz.t = +1.0f;
            mmstep.z *= 12.0f;
        }
        if (pbc.z == 0 && k >= size.z - 2) {
            cfz.x = +1.0f;
            cfz.y = -2.0f;
            cfz.z = +1.0f;
            cfz.w = +0.0f;
            cfz.t = +0.0f;
            mmstep.z *= 12.0f;
        }
        
        /*if (x0 == 100) {
	        printf("msat: %e  ", m_sat);
	        printf("pre: %e  ", pre);
	        printf("A: %e  ", A);
	        printf("alpha: %e  \n", alpha);
	        printf("prel: %e  ", prel);
	    }*/
	    
        float3 m = make_float3(mx[x0], my[x0], mz[x0]);		
        
        // Longitudinal part
        
        float3 h = make_float3(hx[x0], hy[x0], hz[x0]);
        
        float lr = lambda * dotf(h, m); // lambda * (H, m)   
        
        // Transverse part    
         
        
        // Second-order derivative 5-points stencil

        int xb2 = i - 2;
        int xb1 = i - 1;
        int xf1 = i + 1;
        int xf2 = i + 2;

        int yb2 = j - 2;
        int yb1 = j - 1;
        int yf1 = j + 1;
        int yf2 = j + 2; 

        int zb2 = k - 2;
        int zb1 = k - 1;
        int zf1 = k + 1;
        int zf2 = k + 2;

        int4 yi = make_int4(yb2, yb1, yf1, yf2);		  

        xb2 = (pbc.x == 0 && xb2 < 0)? i : xb2; // backward coordinates are negative
        xb1 = (pbc.x == 0 && xb1 < 0)? i : xb1;
        xf1 = (pbc.x == 0 && xf1 >= size.x)? i : xf1;
        xf2 = (pbc.x == 0 && xf2 >= size.x)? i : xf2;
        
        /*if (i == 0)
        {   
           printf("cfx: %e %e %e %e %e\n", cfx.x, cfx.y, cfx.z, cfx.w, cfx.t);
        }*/
        
        yb2 = (lhx == NULL && yb2 < 0)? j : yb2;
        yb1 = (lhx == NULL && yb1 < 0)? j : yb1;
        yf1 = (rhx == NULL && yf1 >= size.y)? j : yf1;
        yf2 = (rhx == NULL && yf2 >= size.y)? j : yf2;

        zb2 = (pbc.z == 0 && zb2 < 0)? k : zb2;
        zb1 = (pbc.z == 0 && zb1 < 0)? k : zb1;
        zf1 = (pbc.z == 0 && zf1 >= size.z)? k : zf1;
        zf2 = (pbc.z == 0 && zf2 >= size.z)? k : zf2;
                
        xb2 = (xb2 >= 0)? xb2 : size.x + xb2;
        xb1 = (xb1 >= 0)? xb1 : size.x + xb1;
        xf1 = (xf1 < size.x)? xf1 : xf1 - size.x;
        xf2 = (xf2 < size.x)? xf2 : xf2 - size.x;

        yb2 = (yb2 >= 0)? yb2 : size.y + yb2;
        yb1 = (yb1 >= 0)? yb1 : size.y + yb1;
        yf1 = (yf1 < size.y)? yf1 : yf1 - size.y;
        yf2 = (yf2 < size.y)? yf2 : yf2 - size.y;

        zb2 = (zb2 >= 0)? zb2 : size.z + zb2;
        zb1 = (zb1 >= 0)? zb1 : size.z + zb1;
        zf1 = (zf1 < size.z)? zf1 : zf1 - size.z;
        zf2 = (zf2 < size.z)? zf2 : zf2 - size.z;
          
        int comm = j * size.z + k;	   
        int4 xn = make_int4(xb2 * size.w + comm, 
				          xb1 * size.w + comm, 
				          xf1 * size.w + comm, 
				          xf2 * size.w + comm); 
				         

        comm = i * size.w + k; 
        int4 yn = make_int4(yb2 * size.z + comm, 
				          yb1 * size.z + comm, 
				          yf1 * size.z + comm, 
				          yf2 * size.z + comm);


        comm = i * size.w + j * size.z;
        int4 zn = make_int4(zb2 + comm, 
				          zb1 + comm, 
				          zf1 + comm, 
				          zf2 + comm);


        // Let's use 5-point stencil in the bulk and 3-point forward/backward at the boundary
        // CUDA does not have vec3 operators like GLSL has, except of .xxx, 

        float4 HH;

        HH.x = (yi.x >= 0 || lhx == NULL) ? hx[yn.x] : lhx[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hx[yn.y] : lhx[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hx[yn.z] : rhx[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hx[yn.w] : rhx[yn.w];
          	    
        float3 dhxdr2 = 	make_float3(mmstep.x * (cfx.x * hx[xn.x] + cfx.y * hx[xn.y] + cfx.z * hx[x0] + cfx.w * hx[xn.z] + cfx.t * hx[xn.w]),
							            mmstep.y * (cfy.x * HH.x     + cfy.y * HH.y     + cfy.z * hx[x0] + cfy.w * HH.z     + cfy.t * HH.w),
							            mmstep.z * (cfz.x * hx[zn.x] + cfz.y * hx[zn.y] + cfz.z * hx[x0] + cfz.w * hx[zn.z] + cfz.t * hx[zn.w]));
							
        HH.x = (yi.x >= 0 || lhx == NULL) ? hy[yn.x] : lhy[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hy[yn.y] : lhy[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hy[yn.z] : rhy[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hy[yn.w] : rhy[yn.w];
						              
        float3 dhydr2 = 	make_float3(mmstep.x * (cfx.x * hy[xn.x] + cfx.y * hy[xn.y] + cfx.z * hy[x0] + cfx.w * hy[xn.z] + cfx.t * hy[xn.w]),
						                mmstep.y * (cfy.x * HH.x     + cfy.y * HH.y     + cfy.z * hy[x0] + cfy.w * HH.z     + cfy.t * HH.w),
							            mmstep.z * (cfz.x * hy[zn.x] + cfz.y * hy[zn.y] + cfz.z * hy[x0] + cfz.w * hy[zn.z] + cfz.t * hy[zn.w]));
							
        HH.x = (yi.x >= 0 || lhx == NULL) ? hz[yn.x] : lhz[yn.x];
        HH.y = (yi.y >= 0 || lhx == NULL) ? hz[yn.y] : lhz[yn.y];
        HH.z = (yi.z < size.y || rhx == NULL) ? hz[yn.z] : rhz[yn.z];
        HH.w = (yi.w < size.y || rhx == NULL) ? hz[yn.w] : rhz[yn.w]; 								
							
								
        float3 dhzdr2 = 	make_float3(mmstep.x * (cfx.x * hz[xn.x] + cfx.y * hz[xn.y] + cfx.z * hz[x0] + cfx.w * hz[xn.z] + cfx.t * hz[xn.w]),
							            mmstep.y * (cfy.x * HH.x     + cfy.y * HH.y     + cfy.z * hz[x0] + cfy.w * HH.z     + cfy.t * HH.w),
						                mmstep.z * (cfz.x * hz[zn.x] + cfz.y * hz[zn.y] + cfz.z * hz[x0] + cfz.w * hz[zn.z] + cfz.t * hz[zn.w])); 


	            
        float3 ddh = make_float3(dhxdr2.x + dhxdr2.y + dhxdr2.z, dhydr2.x + dhydr2.y + dhydr2.z, dhzdr2.x + dhzdr2.y + dhzdr2.z);

		// Longitudinal part
					
	    float le = lambda_e * dotf(m, ddh); // Lambda_e * (m, laplace(h)  
	    l[x0] = (lr - le) / msatMul; // lr - le, since normalize m/As to 1/s, gammaLL is in multiplier
	    
	    //*****************    	  
	    
        float3 ddhxm = crossf(m, ddh); // no minus in it, but it was an interesting behaviour when damping is pumping

        float3 mxddhxm = crossf(m, ddhxm); // with plus from [ddh x m]    
        
        float3 _mxh = crossf(h, m);
        float3 _mxmxh = crossf(m, _mxh);
        
        tx[x0] = _mxh.x + m_sat * (lambda * _mxmxh.x  + lambda_e * mxddhxm.x);
        ty[x0] = _mxh.y + m_sat * (lambda * _mxmxh.y  + lambda_e * mxddhxm.y);
        tz[x0] = _mxh.z + m_sat * (lambda * _mxmxh.z  + lambda_e * mxddhxm.z);  
        

    } 
  }

  
#define BLOCKSIZE 16


  
__export__  void tbaryakhtar_async(float** tx, float**  ty, float**  tz, 
             float** l,
			 float**  mx, float**  my, float**  mz, 
			 float**  hx, float**  hy, float**  hz,
			 float**  msat,
			 const float msatMul, 
			 const float lambda,
			 const float lambda_e,
			 const int sx, const int sy, const int sz,
			 const float csx, const float csy, const float csz,
			 const int pbc_x, const int pbc_y, const int pbc_z, 
			 hipStream_t* stream)
  {

	// 3D :)
	
	dim3 gridSize(divUp(sy, BLOCKSIZE), divUp(sz, BLOCKSIZE));
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
		
	// FUCKING THREADS PER BLOCK LIMITATION
	check3dconf(gridSize, blockSize);
		
	float i12csx = 1.0f / (12.0f * csx * csx);
	float i12csy = 1.0f / (12.0f * csy * csy);
	float i12csz = 1.0f / (12.0f * csz * csz);
	
	int syz = sy * sz;
	
		
	float3 mstep = make_float3(i12csx, i12csy, i12csz);	
	int4 size = make_int4(sx, sy, sz, syz);
	int3 pbc = make_int3(pbc_x, pbc_y, pbc_z);
	
    int nDev = nDevice();
		
	/*hipEvent_t start,stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);*/
		
	
	
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));	 
	  		
		// calculate dev neighbours
		
		int ld = Mod(dev - 1, nDev);
		int rd = Mod(dev + 1, nDev);
				
		float* lhx = hx[ld]; 
		float* lhy = hy[ld];
		float* lhz = hz[ld];

		float* rhx = hx[rd]; 
		float* rhy = hy[rd];
		float* rhz = hz[rd];
		
		if(pbc_y == 0){             
			if(dev == 0){
				lhx = NULL;
				lhy = NULL;
				lhz = NULL;			
			}
			if(dev == nDev-1){
				rhx = NULL;
				rhy = NULL;
				rhz = NULL;
			}
		}
		
		// printf("Devices are: %d\t%d\t%d\n", ld, dev, rd);
		
		for (int i = 0; i < sx; i++) {
												   
			tbaryakhtar_delta2HKernMGPU<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (tx[dev], ty[dev], tz[dev],  
			                                       l[dev],
												   mx[dev], my[dev], mz[dev],												   
												   hx[dev], hy[dev], hz[dev],
												   lhx, lhy, lhz,
												   rhx, rhy, rhz,
												   msat[dev],
												   msatMul,
												   lambda,
												   lambda_e,
												   size,
												   mstep,
												   pbc,
												   i);
		}

    } // end dev < nDev loop
	
	
	/*hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("T-Baryakhtar kernel requires: %f ms\n",time);*/
	
  }
  
  // ========================================

#ifdef __cplusplus
}
#endif
