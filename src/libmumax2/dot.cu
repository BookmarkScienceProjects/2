#include "hip/hip_runtime.h"
#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void dotKern(float* dst,
                        float* ax, float* ay, float* az, 
                        float* bx, float* by, float* bz, 
						int Npart) {
	int i = threadindex;
	if (i < Npart) {
	    float3 a = make_float3(ax[i], ay[i], az[i]);
	    float3 b = make_float3(bx[i], by[i], bz[i]);
        dst[i] = dotf(a, b); 
	}
}

///@internal
__global__ void dotSignKern(float* dst,
                        float* ax, float* ay, float* az, 
                        float* bx, float* by, float* bz, 
			float* cx, float* cy, float* cz,
						int Npart) {
	int i = threadindex;
	if (i < Npart) {
	    float3 a = make_float3(ax[i], ay[i], az[i]);
	    float3 b = make_float3(bx[i], by[i], bz[i]);
	    float3 c = make_float3(cx[i], cy[i], cz[i]);
	    float dotP = dotf(a,b);
	    float sign = dotf(b,c);
        dst[i] = copysign(dotP, sign); 
	}
}


__export__ void dotAsync(float** dst, float** ax, float** ay, float** az, float** bx, float** by, float** bz, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		assert(dst[dev] != NULL);
		assert(ax[dev] != NULL);
		assert(ay[dev] != NULL);
		assert(az[dev] != NULL);
		assert(bx[dev] != NULL);
		assert(by[dev] != NULL);
		assert(bz[dev] != NULL);
		// alphaMap may be null
		gpu_safe(hipSetDevice(deviceId(dev)));
		dotKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], ax[dev],ay[dev],az[dev], bx[dev],by[dev],bz[dev], Npart);
	}
}

__export__ void dotSignAsync(float** dst, float** ax, float** ay, float** az, float** bx, float** by, float** bz, float** cx, float** cy, float** cz, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		assert(dst[dev] != NULL);
		assert(ax[dev] != NULL);
		assert(ay[dev] != NULL);
		assert(az[dev] != NULL);
		assert(bx[dev] != NULL);
		assert(by[dev] != NULL);
		assert(bz[dev] != NULL);
		assert(cx[dev] != NULL);
		assert(cy[dev] != NULL);
		assert(cz[dev] != NULL);
		// alphaMap may be null
		gpu_safe(hipSetDevice(deviceId(dev)));
		dotSignKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], 
										     ax[dev],ay[dev],az[dev], 
										     bx[dev],by[dev],bz[dev],
									             cx[dev],cy[dev],cz[dev],
										     Npart);
	}
}


#ifdef __cplusplus
}
#endif
