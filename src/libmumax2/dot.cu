#include "hip/hip_runtime.h"
#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif

///@internal
    __global__ void dotKern(float* __restrict__ dst,
                            float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
                            float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
                            int Npart)
    {
        int i = threadindex;
        if (i < Npart)
        {
            float3 a = make_float3(ax[i], ay[i], az[i]);
            float3 b = make_float3(bx[i], by[i], bz[i]);
            dst[i] = dotf(a, b);
        }
    }



///@internal
// if b || c, then dst < 0.
    __global__ void dotSignKern(float* __restrict__ dst,
                                float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
                                float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
                                float* __restrict__ cx, float* __restrict__ cy, float* __restrict__ cz,
                                int Npart)
    {
        int i = threadindex;
        if (i < Npart)
        {
            float3 a = make_float3(ax[i], ay[i], az[i]);
            float3 b = make_float3(bx[i], by[i], bz[i]);
            float3 c = make_float3(cx[i], cy[i], cz[i]);
            float dotP = dotf(a, b);
            float sign = -dotf(b, c); // !!!
            dst[i] = copysign(dotP, sign);
        }
    }


    __export__ void dotAsync(float** dst, float** ax, float** ay, float** az, float** bx, float** by, float** bz, hipStream_t* stream, int Npart)
    {
        dim3 gridSize, blockSize;
        make1dconf(Npart, &gridSize, &blockSize);
        for (int dev = 0; dev < nDevice(); dev++)
        {
            assert(dst[dev] != NULL);
            assert(ax[dev] != NULL);
            assert(ay[dev] != NULL);
            assert(az[dev] != NULL);
            assert(bx[dev] != NULL);
            assert(by[dev] != NULL);
            assert(bz[dev] != NULL);
            // alphaMap may be null
            gpu_safe(hipSetDevice(deviceId(dev)));
            dotKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], ax[dev], ay[dev], az[dev], bx[dev], by[dev], bz[dev], Npart);
        }
    }

    __export__ void dotSignAsync(float** dst, float** ax, float** ay, float** az, float** bx, float** by, float** bz, float** cx, float** cy, float** cz, hipStream_t* stream, int Npart)
    {
        dim3 gridSize, blockSize;
        make1dconf(Npart, &gridSize, &blockSize);
        for (int dev = 0; dev < nDevice(); dev++)
        {
            assert(dst[dev] != NULL);
            assert(ax[dev] != NULL);
            assert(ay[dev] != NULL);
            assert(az[dev] != NULL);
            assert(bx[dev] != NULL);
            assert(by[dev] != NULL);
            assert(bz[dev] != NULL);
            assert(cx[dev] != NULL);
            assert(cy[dev] != NULL);
            assert(cz[dev] != NULL);
            // alphaMap may be null
            gpu_safe(hipSetDevice(deviceId(dev)));
            dotSignKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev],
                    ax[dev], ay[dev], az[dev],
                    bx[dev], by[dev], bz[dev],
                    cx[dev], cy[dev], cz[dev],
                    Npart);
        }
    }


#ifdef __cplusplus
}
#endif
