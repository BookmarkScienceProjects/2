#include "hip/hip_runtime.h"
#include "slonczewski_torque.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include <hip/hip_runtime.h>
#include "common_func.h"


#ifdef __cplusplus
extern "C" {
#endif
  // ========================================

  __global__ void slonczewski_deltaMKern(float* __restrict__ sttx, float* __restrict__ stty, float* __restrict__ sttz, 
					 float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz, 
					 float* __restrict__ msat,
					 float* __restrict__ px, float* __restrict__ py, float* __restrict__ pz,
					 float* __restrict__ jx, float* __restrict__ jy, float* __restrict__ jz,
					 float* __restrict__ alphaMsk, 
					 float3 pMul,
					 float3 jMul,
					 float3 pre,
					 float3 meshSize,
					 float alphaMul,
					 int NPart) 
  {
    
    int I = threadindex;
    float Ms = (msat != NULL ) ? msat[I] : 1.0f;
    
    if (Ms == 0.0f) {
      sttx[I] = 0.0f;
      stty[I] = 0.0f;
      sttz[I] = 0.0f;    
      return;
    }
    
    float j_x = (jx != NULL) ? jx[I] * jMul.x : jMul.x;
    float j_y = (jy != NULL) ? jy[I] * jMul.y : jMul.y;
    float j_z = (jz != NULL) ? jz[I] * jMul.z : jMul.z;

    float3 J = make_float3(j_x, j_y, j_z);
    float nJn = len(J);
    
    if (nJn == 0.0f) {
      sttx[I] = 0.0f;
      stty[I] = 0.0f;
      sttz[I] = 0.0f;    
      return;  
    }
	  
	if (I < NPart){ // Thread configurations are usually too large...

      Ms = 1.0f / Ms;
          
      pre.y *= Ms;
      pre.z *= Ms;
       
      float3 m = make_float3(mx[I], my[I], mz[I]);
      
      float p_x = (px != NULL) ? pMul.x * px[I] : pMul.x;
      float p_y = (py != NULL) ? pMul.y * py[I] : pMul.y;
      float p_z = (pz != NULL) ? pMul.z * pz[I] : pMul.z;  
        
      float3 p = make_float3(p_x, p_y, p_z);  
                   
      p = normalize(p);
       
      float3 pxm = crossf(p, m); // plus
      float3 mxpxm = crossf(m, pxm); // plus 
      
      float  pdotm = dotf(p, m);
           
	  J = normalize(J);
	  float Jdir = dotf(make_float3(1.0f,1.0f,1.0f), J);
	  float Jsign = Jdir / fabsf(Jdir); 
	  nJn *= Jsign; 
	  pre.y *= nJn;
	  pre.z *= nJn;
	  
	  // get effective thinkness of free layer
	  
	  float free_layer_thickness = fabsf(dotf(meshSize, J)); 
	  free_layer_thickness = (free_layer_thickness != 0.0f) ? 1.0f / free_layer_thickness : 0.0f;
	  pre.y *= free_layer_thickness;
	  pre.z *= free_layer_thickness; 
	  
      float epsilon = pre.x / ((pre.x + 1.0f) + (pre.x - 1.0f) * pdotm);
      pre.y *= epsilon;
      
      float alpha = (alphaMsk != NULL) ? 1.0f/(1.0f + alphaMsk[I] * alphaMul * alphaMsk[I] * alphaMul) : 1.0f/(1.0f + alphaMul * alphaMul); 
      pre.y *= alpha;
      pre.z *= alpha;
     
      sttx[I] = pre.y * mxpxm.x + pre.z * pxm.x;
      stty[I] = pre.y * mxpxm.y + pre.z * pxm.y;
      sttz[I] = pre.y * mxpxm.z + pre.z * pxm.z;
      
    } 
  }

  #define BLOCKSIZE 16
  
  void slonczewski_async(float** sttx, float** stty, float** sttz, 
			 float** mx, float** my, float** mz, 
			 float** msat,
			 float** px, float** py, float** pz,
			 float** jx, float** jy, float** jz,
			 float** alphamsk,
			 float pxMul, float pyMul, float pzMul,
			 float jxMul, float jyMul, float jzMul,
			 float lambda2, float beta_prime, float pre_field,
			 float meshSizeX,float meshSizeY, float meshSizeZ, 
			 float alphaMul,
			 int NPart, 
			 hipStream_t* stream)
  {

    // 1D configuration
    dim3 gridSize, blockSize;
    make1dconf(NPart, &gridSize, &blockSize);
    float3 meshSize = make_float3(meshSizeX, meshSizeY, meshSizeZ);
    float3 pre = make_float3(lambda2, beta_prime, pre_field);
    float3 pMul = make_float3(pxMul, pyMul, pzMul);
    float3 jMul = make_float3(jxMul, jyMul, jzMul);
    
    int nDev = nDevice();
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));
	    slonczewski_deltaMKern<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (sttx[dev], stty[dev], sttz[dev],  
										       mx[dev], my[dev], mz[dev],
										       msat[dev],  
										       px[dev], py[dev], pz[dev],
										       jx[dev], jy[dev], jz[dev],
										       alphamsk[dev],
											   pMul,
											   jMul,
											   pre,
										       meshSize,
										       alphaMul, 
										       NPart);
    } // end dev < nDev loop
										  
										  
  }

  // ========================================

#ifdef __cplusplus
}
#endif
