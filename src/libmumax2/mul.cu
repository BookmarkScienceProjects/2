#include "hip/hip_runtime.h"
#include "mul.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void mulKern(float* dst, float* a, float* b, int Npart)
{
    int i = threadindex;
    if (i < Npart)
    {
        dst[i] = a[i] * b[i];
    }
}


__export__ void mulAsync(float** dst, float** a, float** b, hipStream_t* stream, int Npart)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    for (int dev = 0; dev < nDevice(); dev++)
    {
        gpu_safe(hipSetDevice(deviceId(dev)));
        mulKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (dst[dev], a[dev], b[dev], Npart);
    }
}

#ifdef __cplusplus
}
#endif

