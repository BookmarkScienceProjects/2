#include "hip/hip_runtime.h"

#include "add.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void addKern(float *dst, float *a, float *b, int Npart) {
	int i = threadindex;
	if (i < Npart) {
		dst[i] = a[i] + b[i];
	}
}


void addAsync(float **dst, float **a, float **b, hipStream_t * stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int i = 0; i < nDevice(); i++) {
		gpu_safe(hipSetDevice(deviceId(i)));
		addKern <<< gridSize, blockSize, 0, hipStream_t(stream[i]) >>> (dst[i], a[i], b[i], Npart);
	}
}

#ifdef __cplusplus
}
#endif
