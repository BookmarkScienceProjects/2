#include "hip/hip_runtime.h"
#include "uniaxialanisotropy.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

__global__ void uniaxialAnisotropyKern (float *hx, float *hy, float *hz, 
                                     float *mx, float *my, float *mz,
                                     float *Ku1_map, float Ku1_mul, 
                                     float *Ku2_map, float Ku2_mul, 
                                     float *anisU_mapx, float anisU_mulx,
                                     float *anisU_mapy, float anisU_muly,
                                     float *anisU_mapz, float anisU_mulz,
                                     int Npart){

  int i = threadindex;

  if (i < Npart){

    float Ku1;
    if (Ku1_map==NULL){
      Ku1 = Ku1_mul;
	}else{
      Ku1 = Ku1_mul*Ku1_map[i];
	}

    float Ku2;
    if (Ku2_map==NULL){
      Ku2 = Ku2_mul;
	}else{
      Ku2 = Ku2_mul*Ku2_map[i];
	}

    float ux;
    if (anisU_mapx==NULL){
      ux = anisU_mulx;
    }else{
      ux = anisU_mulx*anisU_mapx[i];
    }
    
    float uy;
    if (anisU_mapy==NULL){
      uy = anisU_muly;
    }else{
      uy = anisU_muly*anisU_mapy[i];
    }
    
    float uz;
    if (anisU_mapz==NULL){
      uz = anisU_mulz;
    }else{
      uz = anisU_mulz*anisU_mapz[i];
    }
    
    float mu = mx[i]*ux + my[i]*uy + mz[i]*uz;
    hx[i] = Ku1*mu*ux;
    hy[i] = Ku1*mu*uy;
    hz[i] = Ku1*mu*uz;
  }

}



void uniaxialAnisotropyAsync(float **hx, float **hy, float **hz, 
                          float **mx, float **my, float **mz,
                          float **Ku1_map, float Ku1_mul, 
                          float **Ku2_map, float Ku2_mul, 
                          float **anisU_mapx, float anisU_mulx,
                          float **anisU_mapy, float anisU_muly,
                          float **anisU_mapz, float anisU_mulz,
                          hipStream_t* stream, int Npart){

  assert(Ku2_mul == 0); // todo: implement 2nd order
  dim3 gridSize, blockSize;
  make1dconf(Npart, &gridSize, &blockSize);

  for (int dev=0; dev<nDevice(); dev++){
    assert(hx[dev] != NULL);
    assert(hy[dev] != NULL);
    assert(hz[dev] != NULL);
    assert(mx[dev] != NULL);
    assert(my[dev] != NULL);
    assert(mz[dev] != NULL);
    gpu_safe(hipSetDevice(deviceId(dev)));

    uniaxialAnisotropyKern<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (
					hx[dev],hy[dev],hz[dev],  
                    mx[dev],my[dev],mz[dev], 
                    Ku1_map[dev], Ku1_mul,
                    Ku2_map[dev], Ku2_mul,
                    anisU_mapx[dev], anisU_mulx,
                    anisU_mapy[dev], anisU_muly,
                    anisU_mapz[dev], anisU_mulz,
                    Npart);
  }
}

#ifdef __cplusplus
}
#endif
