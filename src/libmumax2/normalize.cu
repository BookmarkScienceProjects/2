#include "hip/hip_runtime.h"
#include "normalize.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void normalizeKern(float* mx, float* my, float* mz,
                              int Npart)
{
    int i = threadindex;

    if (i < Npart)
    {
        // reconstruct norm from map

        float Mx = mx[i];
        float My = my[i];
        float Mz = mz[i];

        float Mnorm = 1.0f / sqrtf(Mx * Mx + My * My + Mz * Mz);

        mx[i] = Mx * Mnorm;
        my[i] = My * Mnorm;
        mz[i] = Mz * Mnorm;
    }
}


__export__ void normalizeAsync(float** mx, float** my, float** mz, hipStream_t* stream, int Npart)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    for (int dev = 0; dev < nDevice(); dev++)
    {
        assert(mx[dev] != NULL);
        assert(my[dev] != NULL);
        assert(mz[dev] != NULL);
        // normMap may be null
        gpu_safe(hipSetDevice(deviceId(dev)));
        normalizeKern <<< gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (mx[dev], my[dev], mz[dev], Npart);
    }
}

#ifdef __cplusplus
}
#endif
