#include "hip/hip_runtime.h"
#include "long_field.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include <hip/hip_runtime.h>
#include "common_func.h"


#ifdef __cplusplus
extern "C" {
#endif
  // ========================================

  __global__ void long_field_Kern(float* __restrict__ hx, float* __restrict__ hy, float* __restrict__ hz, 
					 float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                     float* __restrict__ msatMsk, 
					 float* __restrict__ msat0Msk,
					 float* __restrict__ kappaMsk,
					 float kappaMul,
					 float msatMul,
					 float msat0Mul,
					 int NPart) 
  {
    
    int I = threadindex;
    real Ms0 = (msat0Msk != NULL ) ? msat0Msk[I] * msat0Mul : msat0Mul;
    real kappa = (kappaMsk != NULL ) ? kappaMsk[I] * kappaMul : kappaMul;
    
    if (Ms0 == 0.0 || kappa == 0.0) {
      hx[I] = 0.0f;
      hy[I] = 0.0f;
      hz[I] = 0.0f;    
      return;
    }
    
    if (I < NPart){ // Thread configurations are usually too large...
      
      kappa = 1.0 / kappa;
      
      real Ms = (msatMsk != NULL ) ? msatMsk[I] * msatMul : msatMul;
      
      real3 m = make_real3(mx[I], my[I], mz[I]);
      
      real ratio = Ms/Ms0;
       
      real mult = Ms * kappa * (1.0 - ratio * ratio);// kappa is actually 0.5/kappa! 
         
      hx[I] = mult * m.x;
      hy[I] = mult * m.y;
      hz[I] = mult * m.z;      
    } 
  }

  #define BLOCKSIZE 16
  
  void long_field_async(float** hx, float** hy, float** hz, 
			 float** mx, float** my, float** mz,
			 float** msat, 
			 float** msat0,
			 float** kappa, 
			 float kappaMul,
			 float msatMul, 
			 float msat0Mul,    
			 int NPart,     
			 hipStream_t* stream)
  {

    // 1D configuration
    dim3 gridSize, blockSize;
    make1dconf(NPart, &gridSize, &blockSize);
    
    int nDev = nDevice();
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));
	    long_field_Kern<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (hx[dev], hy[dev], hz[dev],  
										       mx[dev], my[dev], mz[dev],
										       msat[dev], 
										       msat0[dev],
										       kappa[dev], 
										       kappaMul,
										       msatMul, 
										       msat0Mul,
										       NPart);
    } // end dev < nDev loop
										    
										  
  }

  // ========================================

#ifdef __cplusplus
}
#endif
