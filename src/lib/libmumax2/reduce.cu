#include "hip/hip_runtime.h"
// The code in this source file is based on the reduction code from the CUDPP library. Hence the following notice:

/*
Copyright (c) 2007-2010 The Regents of the University of California, Davis
campus ("The Regents") and NVIDIA Corporation ("NVIDIA"). All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright notice,
      this list of conditions and the following disclaimer in the documentation
      and/or other materials provided with the distribution.
    * Neither the name of the The Regents, nor NVIDIA, nor the names of its
      contributors may be used to endorse or promote products derived from this
      software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// This code has been significantly modified from its original version by Arne Vansteenkiste, 2011.
//  - restricted to use only floats
//  - more reduction operations than the original "sum" have been added (min, max, maxabs, ...)
//  - added streams for asynchronous execution
// Note that you have to comply with both the above BSD and GPL licences.

//  This file is part of MuMax, a high-performance micromagnetic simulator.
//  Copyright 2011  Arne Vansteenkiste and Ben Van de Wiele.
//  Use of this source code is governed by the GNU General Public License version 3
//  (as published by the Free Software Foundation) that can be found in the license.txt file.
//  Note that you are welcome to modify this code under the condition that you do not remove any 
//  copyright notices and prominently state that you modified it, giving a relevant date.

///@todo case 1024 should be added to take advantage of modern GPUs

#include "reduce.h"
#include "multigpu.h"
#include "gpu_safe.h"

extern "C"
bool isPow2(unsigned int x){
  return ((x&(x-1))==0);
}

/// @internal
/// Utility class used to avoid linker errors with extern
/// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory {
  __device__ inline operator       T*()
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }

  __device__ inline operator const T*() const
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }
};

//________________________________________________________________________________________________________________ kernels


/// This kernel takes a partial sum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_sum_kernel(float* g_idata, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    mySum += g_idata[i];
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      mySum += g_idata[i+blockSize];
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  __syncthreads();


  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { mySum = mySum + sdata[tid + 256]; sdata[tid] = mySum; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { mySum = mySum + sdata[tid + 128]; sdata[tid] = mySum; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { mySum = mySum + sdata[tid +  64]; sdata[tid] = mySum; } __syncthreads(); }

  if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { mySum = mySum + smem[tid + 32]; smem[tid] = mySum;  }
      if (blockSize >=  32) { mySum = mySum + smem[tid + 16]; smem[tid] = mySum;  }
      if (blockSize >=  16) { mySum = mySum + smem[tid +  8]; smem[tid] = mySum;  }
      if (blockSize >=   8) { mySum = mySum + smem[tid +  4]; smem[tid] = mySum;  }
      if (blockSize >=   4) { mySum = mySum + smem[tid +  2]; smem[tid] = mySum;  }
      if (blockSize >=   2) { mySum = mySum + smem[tid +  1]; smem[tid] = mySum;  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}

/// This kernel takes a partial maximum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_max_kernel(float* g_idata, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float myMax = -6E38;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    myMax = fmax(myMax, g_idata[i]);
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      myMax = fmax(myMax, g_idata[i+blockSize]);
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = myMax;
  __syncthreads();


  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { myMax = fmax(myMax, sdata[tid + 256]); sdata[tid] = myMax; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { myMax = fmax(myMax, sdata[tid + 128]); sdata[tid] = myMax; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { myMax = fmax(myMax, sdata[tid +  64]); sdata[tid] = myMax; } __syncthreads(); }

  if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { myMax = fmax(myMax, smem[tid + 32]); smem[tid] = myMax;  }
      if (blockSize >=  32) { myMax = fmax(myMax, smem[tid + 16]); smem[tid] = myMax;  }
      if (blockSize >=  16) { myMax = fmax(myMax, smem[tid +  8]); smem[tid] = myMax;  }
      if (blockSize >=   8) { myMax = fmax(myMax, smem[tid +  4]); smem[tid] = myMax;  }
      if (blockSize >=   4) { myMax = fmax(myMax, smem[tid +  2]); smem[tid] = myMax;  }
      if (blockSize >=   2) { myMax = fmax(myMax, smem[tid +  1]); smem[tid] = myMax;  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial minimum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_min_kernel(float* g_idata, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float myMin = 6E38;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    myMin = fmin(myMin, g_idata[i]);
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      myMin = fmin(myMin, g_idata[i+blockSize]);
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = myMin;
  __syncthreads();


  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { myMin = fmin(myMin, sdata[tid + 256]); sdata[tid] = myMin; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { myMin = fmin(myMin, sdata[tid + 128]); sdata[tid] = myMin; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { myMin = fmin(myMin, sdata[tid +  64]); sdata[tid] = myMin; } __syncthreads(); }

  if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { myMin = fmin(myMin, smem[tid + 32]); smem[tid] = myMin;  }
      if (blockSize >=  32) { myMin = fmin(myMin, smem[tid + 16]); smem[tid] = myMin;  }
      if (blockSize >=  16) { myMin = fmin(myMin, smem[tid +  8]); smem[tid] = myMin;  }
      if (blockSize >=   8) { myMin = fmin(myMin, smem[tid +  4]); smem[tid] = myMin;  }
      if (blockSize >=   4) { myMin = fmin(myMin, smem[tid +  2]); smem[tid] = myMin;  }
      if (blockSize >=   2) { myMin = fmin(myMin, smem[tid +  1]); smem[tid] = myMin;  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial maximum of absolute values
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxabs_kernel(float* g_idata, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float myMaxabs = 0.;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    myMaxabs = fmax(myMaxabs, fabs(g_idata[i]));
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      myMaxabs = fmax(myMaxabs, fabs(g_idata[i+blockSize]));
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = myMaxabs;
  __syncthreads();


  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { myMaxabs = fmax(myMaxabs, sdata[tid + 256]); sdata[tid] = myMaxabs; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { myMaxabs = fmax(myMaxabs, sdata[tid + 128]); sdata[tid] = myMaxabs; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { myMaxabs = fmax(myMaxabs, sdata[tid +  64]); sdata[tid] = myMaxabs; } __syncthreads(); }

  if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { myMaxabs = fmax(myMaxabs, smem[tid + 32]); smem[tid] = myMaxabs;  }
      if (blockSize >=  32) { myMaxabs = fmax(myMaxabs, smem[tid + 16]); smem[tid] = myMaxabs;  }
      if (blockSize >=  16) { myMaxabs = fmax(myMaxabs, smem[tid +  8]); smem[tid] = myMaxabs;  }
      if (blockSize >=   8) { myMaxabs = fmax(myMaxabs, smem[tid +  4]); smem[tid] = myMaxabs;  }
      if (blockSize >=   4) { myMaxabs = fmax(myMaxabs, smem[tid +  2]); smem[tid] = myMaxabs;  }
      if (blockSize >=   2) { myMaxabs = fmax(myMaxabs, smem[tid +  1]); smem[tid] = myMaxabs;  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}



/// This kernel takes a partial maximum difference between two arrays
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxdiff_kernel(float* a, float* b, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  float myMaxabs = 0.;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    myMaxabs = fmax(myMaxabs, fabs(a[i]-b[i]));
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      myMaxabs = fmax(myMaxabs, fabs(a[i+blockSize]-b[i+blockSize]));
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = myMaxabs;
  __syncthreads();


  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { myMaxabs = fmax(myMaxabs, sdata[tid + 256]); sdata[tid] = myMaxabs; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { myMaxabs = fmax(myMaxabs, sdata[tid + 128]); sdata[tid] = myMaxabs; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { myMaxabs = fmax(myMaxabs, sdata[tid +  64]); sdata[tid] = myMaxabs; } __syncthreads(); }

  if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { myMaxabs = fmax(myMaxabs, smem[tid + 32]); smem[tid] = myMaxabs;  }
      if (blockSize >=  32) { myMaxabs = fmax(myMaxabs, smem[tid + 16]); smem[tid] = myMaxabs;  }
      if (blockSize >=  16) { myMaxabs = fmax(myMaxabs, smem[tid +  8]); smem[tid] = myMaxabs;  }
      if (blockSize >=   8) { myMaxabs = fmax(myMaxabs, smem[tid +  4]); smem[tid] = myMaxabs;  }
      if (blockSize >=   4) { myMaxabs = fmax(myMaxabs, smem[tid +  2]); smem[tid] = myMaxabs;  }
      if (blockSize >=   2) { myMaxabs = fmax(myMaxabs, smem[tid +  1]); smem[tid] = myMaxabs;  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}

//________________________________________________________________________________________________________________ kernel wrappers

#ifdef __cplusplus
extern "C" {
#endif

// single-GPU
void partialSumAsync1(float* d_idata, float* d_odata, int blocks, int threads, int size, hipStream_t stream) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  if (isPow2(size))
  {
    switch (threads)
    {
      case 512: _gpu_sum_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_sum_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_sum_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_sum_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_sum_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_sum_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_sum_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_sum_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_sum_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_sum_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
  else
  {
    switch (threads)
    {
      case 512: _gpu_sum_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_sum_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_sum_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_sum_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_sum_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_sum_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_sum_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_sum_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_sum_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_sum_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
}

void partialSumAsync(float** input, float** output, int blocks, int threadsPerBlock, int N, hipStream_t* stream){
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		partialSumAsync1(input[dev], output[dev], blocks, threadsPerBlock, N, stream[dev]);
	}
}





// single-GPU
void partialMaxAsync1(float* d_idata, float* d_odata, int blocks, int threads, int size, hipStream_t stream) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  if (isPow2(size))
  {
    switch (threads)
    {
      case 512: _gpu_max_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_max_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_max_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_max_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_max_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_max_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_max_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_max_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_max_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_max_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
  else
  {
    switch (threads)
    {
      case 512: _gpu_max_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_max_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_max_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_max_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_max_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_max_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_max_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_max_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_max_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_max_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
}


void partialMaxAsync(float** input, float** output, int blocks, int threadsPerBlock, int N, hipStream_t* stream){
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		partialMaxAsync1(input[dev], output[dev], blocks, threadsPerBlock, N, stream[dev]);
	}
}





void partialMinAsync1(float* d_idata, float* d_odata, int blocks, int threads, int size, hipStream_t stream) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  if (isPow2(size))
  {
    switch (threads)
    {
      case 512: _gpu_min_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_min_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_min_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_min_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_min_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_min_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_min_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_min_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_min_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_min_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
  else
  {
    switch (threads)
    {
      case 512: _gpu_min_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_min_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_min_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_min_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_min_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_min_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_min_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_min_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_min_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_min_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
}


void partialMinAsync(float** input, float** output, int blocks, int threadsPerBlock, int N, hipStream_t* stream){
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		partialMinAsync1(input[dev], output[dev], blocks, threadsPerBlock, N, stream[dev]);
	}
}




// Single-GPU
void partialMaxAbsAsync1(float* d_idata, float* d_odata, int blocks, int threads, int size, hipStream_t stream) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  if (isPow2(size))
  {
    switch (threads)
    {
      case 512: _gpu_maxabs_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_maxabs_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_maxabs_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_maxabs_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_maxabs_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_maxabs_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_maxabs_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_maxabs_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_maxabs_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_maxabs_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
  else
  {
    switch (threads)
    {
      case 512: _gpu_maxabs_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 256: _gpu_maxabs_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case 128: _gpu_maxabs_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  64: _gpu_maxabs_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  32: _gpu_maxabs_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case  16: _gpu_maxabs_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   8: _gpu_maxabs_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   4: _gpu_maxabs_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   2: _gpu_maxabs_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
      case   1: _gpu_maxabs_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
    }
  }
}


void partialMaxAbsAsync(float** input, float** output, int blocks, int threadsPerBlock, int N, hipStream_t* stream){
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		partialMaxAbsAsync1(input[dev], output[dev], blocks, threadsPerBlock, N, stream[dev]);
	}
}



// Single-GPU
void partialMaxDiffAsync1(float* a, float* b, float* d_odata, int blocks, int threads, int size, hipStream_t stream) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  if (isPow2(size))
  {
    switch (threads)
    {
      case 512: _gpu_maxdiff_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case 256: _gpu_maxdiff_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case 128: _gpu_maxdiff_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  64: _gpu_maxdiff_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  32: _gpu_maxdiff_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  16: _gpu_maxdiff_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   8: _gpu_maxdiff_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   4: _gpu_maxdiff_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   2: _gpu_maxdiff_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   1: _gpu_maxdiff_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
    }
  }
  else
  {
    switch (threads)
    {
      case 512: _gpu_maxdiff_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case 256: _gpu_maxdiff_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case 128: _gpu_maxdiff_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  64: _gpu_maxdiff_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  32: _gpu_maxdiff_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case  16: _gpu_maxdiff_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   8: _gpu_maxdiff_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   4: _gpu_maxdiff_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   2: _gpu_maxdiff_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
      case   1: _gpu_maxdiff_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size); break;
    }
  }
}


void partialMaxDiffAsync(float** a, float** b, float** output, int blocks, int threadsPerBlock, int N, hipStream_t* stream){
	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		partialMaxDiffAsync1(a[dev], b[dev], output[dev], blocks, threadsPerBlock, N, stream[dev]);
	}
}

///// This kernel takes a partial sum of absolute values
//template <unsigned int blockSize, bool nIsPow2>
//__global__ void _gpu_sumabs_kernel(float* g_idata, float* g_odata, unsigned int n) {
//  float* sdata = SharedMemory<float>();
//
//  // perform first level of reduction,
//  // reading from global memory, writing to shared memory
//  unsigned int tid = threadIdx.x;
//  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
//  unsigned int gridSize = blockSize*2*gridDim.x;
//
//  float mySum = 0;
//
//  // we reduce multiple elements per thread.  The number is determined by the
//  // number of active thread blocks (via gridDim).  More blocks will result
//  // in a larger gridSize and therefore fewer elements per thread
//  while (i < n)
//  {
//    mySum += fabs(g_idata[i]);
//    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
//    if (nIsPow2 || i + blockSize < n)
//      mySum += fabs(g_idata[i+blockSize]);
//    i += gridSize;
//  }
//
//  // each thread puts its local sum into shared memory
//  sdata[tid] = mySum;
//  __syncthreads();
//
//
//  // do reduction in shared mem
//  if (blockSize >= 512) { if (tid < 256) { mySum = mySum + sdata[tid + 256]; sdata[tid] = mySum; } __syncthreads(); }
//  if (blockSize >= 256) { if (tid < 128) { mySum = mySum + sdata[tid + 128]; sdata[tid] = mySum; } __syncthreads(); }
//  if (blockSize >= 128) { if (tid <  64) { mySum = mySum + sdata[tid +  64]; sdata[tid] = mySum; } __syncthreads(); }
//
//  if (tid < 32)
//    {
//      // now that we are using warp-synchronous programming (below)
//      // we need to declare our shared memory volatile so that the compiler
//      // doesn't reorder stores to it and induce incorrect behavior.
//      volatile float* smem = sdata;
//      if (blockSize >=  64) { mySum = mySum + smem[tid + 32]; smem[tid] = mySum;  }
//      if (blockSize >=  32) { mySum = mySum + smem[tid + 16]; smem[tid] = mySum;  }
//      if (blockSize >=  16) { mySum = mySum + smem[tid +  8]; smem[tid] = mySum;  }
//      if (blockSize >=   8) { mySum = mySum + smem[tid +  4]; smem[tid] = mySum;  }
//      if (blockSize >=   4) { mySum = mySum + smem[tid +  2]; smem[tid] = mySum;  }
//      if (blockSize >=   2) { mySum = mySum + smem[tid +  1]; smem[tid] = mySum;  }
//    }
//    // write result for this block to global mem
//    if (tid == 0)
//      g_odata[blockIdx.x] = sdata[0];
//}
//
//void gpu_partial_sumabs(float* d_idata, float* d_odata, int blocks, int threads, int size) {
//  dim3 dimBlock(threads, 1, 1);
//  dim3 dimGrid(blocks, 1, 1);
//
//  // when there is only one warp per block, we need to allocate two warps
//  // worth of shared memory so that we don't index shared memory out of bounds
//  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
//
//  if (isPow2(size))
//  {
//    switch (threads)
//    {
//      case 512: _gpu_sumabs_kernel<512, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case 256: _gpu_sumabs_kernel<256, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case 128: _gpu_sumabs_kernel<128, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  64: _gpu_sumabs_kernel< 64, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  32: _gpu_sumabs_kernel< 32, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  16: _gpu_sumabs_kernel< 16, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   8: _gpu_sumabs_kernel<  8, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   4: _gpu_sumabs_kernel<  4, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   2: _gpu_sumabs_kernel<  2, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   1: _gpu_sumabs_kernel<  1, true><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//    }
//  }
//  else
//  {
//    switch (threads)
//    {
//      case 512: _gpu_sumabs_kernel<512, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case 256: _gpu_sumabs_kernel<256, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case 128: _gpu_sumabs_kernel<128, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  64: _gpu_sumabs_kernel< 64, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  32: _gpu_sumabs_kernel< 32, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case  16: _gpu_sumabs_kernel< 16, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   8: _gpu_sumabs_kernel<  8, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   4: _gpu_sumabs_kernel<  4, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   2: _gpu_sumabs_kernel<  2, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//      case   1: _gpu_sumabs_kernel<  1, false><<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size); break;
//    }
//  }
//}


#ifdef __cplusplus
}
#endif
