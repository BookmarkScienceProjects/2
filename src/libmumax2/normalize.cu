#include "hip/hip_runtime.h"
#include "normalize.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void normalizeKern(float* mx, float* my, float* mz, 
						   float* norm_map, int Npart) {
	int i = threadindex;
	if (i < Npart) {

		// reconstruct norm from map
		float norm;
		if(norm_map == NULL){
			norm = 1.0f;
		}else{
			norm = norm_map[i];
		}

    	float Mx = mx[i];
    	float My = my[i];
    	float Mz = mz[i];
    
		float Mnorm = sqrtf(Mx*Mx + My*My + Mz*Mz);
		float scale;
		if (Mnorm != 0.f){
			scale = norm / Mnorm;
			scale = 1.0f / Mnorm;
		}else{
			scale = 0.f;
		}

		mx[i] = Mx * scale;
		my[i] = My * scale;
		mz[i] = Mz * scale;
	}
}


__export__ void normalizeAsync(float** mx, float** my, float** mz, float** norm_map, hipStream_t* stream, int Npart) {
	dim3 gridSize, blockSize;
	make1dconf(Npart, &gridSize, &blockSize);
	for (int dev = 0; dev < nDevice(); dev++) {
		assert(mx[dev] != NULL);
		assert(my[dev] != NULL);
		assert(mz[dev] != NULL);
		// normMap may be null
		gpu_safe(hipSetDevice(deviceId(dev)));
		normalizeKern <<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (mx[dev],my[dev],mz[dev], norm_map[dev], Npart);
	}
}

#ifdef __cplusplus
}
#endif
