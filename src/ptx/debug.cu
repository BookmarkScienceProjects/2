#include "hip/hip_runtime.h"
//  This file is part of MuMax, a high-performance micromagnetic simulator.
//  Copyright 2011  Arne Vansteenkiste and Ben Van de Wiele.
//  Use of this source code is governed by the GNU General Public License version 3
//  (as published by the Free Software Foundation) that can be found in the license.txt file.
//  Note that you are welcome to modify this code under the condition that you do not remove any 
//  copyright notices and prominently state that you modified it, giving a relevant date.

/// This file implements various functions used for debugging.

#include "macros.h"

#ifdef __cplusplus
extern "C" {
#endif

/// @debug sets array[i] to i.
__global__ void SetIndex1D(float* part, int PART, int N){

  int i = threadindex;
  if (i < N){
	part[i] = i + PART*N;
  }
}



/// @debug sets array[i,j,k] to its C-oder index.
__global__ void SetIndex3D(float* part, int PART, int N0, int N1, int N2){

  int k = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N1 && k < N2){
	for(int i=0; i<N0; i++){
  		int I = i*N1*N2 + j*N2 + k; // linear array index
			j += PART * N1;
			part[I] = i*1000 + j + k/1000;
		}
	}
}


#ifdef __cplusplus
}
#endif
