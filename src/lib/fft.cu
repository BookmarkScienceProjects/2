#include "hip/hip_runtime.h"
#include "fft.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif


/// @internal Does Z-padding and unpadding of a 2D matrix.
__global__ void copyPad2dKern(float* dst, int D2, float* src, int S1, int S2){
  
   int i = blockIdx.y * blockDim.y + threadIdx.y;
   int j = blockIdx.x * blockDim.x + threadIdx.x;

	// this check makes it work for padding as well as for unpadding.
	// 2 separate functions are probably not more efficient
	// due to memory bandwidth limitations
   if (i<S1 && j<S2 && j<D2){  // && i<D1: always true
		dst[i*D2 + j] = src[i*S2 + j];
   }
}

void copyPadZAsync(float** dst, int D2, float** src, int S0, int S1Part, int S2, hipStream_t* streams){
	assert(S2 <= D2);

#define BLOCKSIZE 16 ///@todo use device properties
  dim3 gridSize(divUp(S2, BLOCKSIZE), divUp(S1Part, BLOCKSIZE), 1);
  dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
  check3dconf(gridSize, blockSize);

	for (int dev = 0; dev < nDevice(); dev++) {
		gpu_safe(hipSetDevice(deviceId(dev)));
		//for  
			//&source[i*S1*S2],
		copyPad2dKern <<<gridSize, blockSize, 0, hipStream_t(streams[dev])>>> (dst[dev], D2, src[dev], S1Part, S2);
	}
}



#ifdef __cplusplus
}
#endif

