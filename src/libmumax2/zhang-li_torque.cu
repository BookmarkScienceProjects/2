#include "hip/hip_runtime.h"
#include "zhang-li_torque.h"
#include "multigpu.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>

#ifdef __cplusplus
extern "C" {
#endif

	// dot product
	inline __host__ __device__ float dotf(float3 a, float3 b)
	{ 
		return a.x * b.x + a.y * b.y + a.z * b.z;
	}

	// cross product
	inline __host__ __device__ float3 crossf(float3 a, float3 b)
	{ 
		return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x); 
	}
	
  // ========================================

  __global__ void zhangli_deltaMKern(float* sttx, float* stty, float* sttz, 
					 float* mx, float* my, float* mz,					 
					 float* jx, float* jy, float* jz,
					 float* msat,
					 float2 pre,
					 int4 size,		
					 float3 mstep,
					 int i)
  {	
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;			
	int x0 = i * size.w + j * size.z + k;
	
	float m_sat = (msat != NULL) ? msat[x0] : 1.0f;
	
    if (m_sat != 0.0f && j < size.y && k < size.z){ // 3D now:)
	   
	   m_sat = 1.0f / m_sat;
	  
	  
	  float3 m = make_float3(mx[x0], my[x0], mz[x0]);		
		 	
 
      // First-order derivative 5-points stencil
	   
	  int xb2 = (i-2 >= 0)? i-2 : i;
	  int xb1 = (i-1 >= 0)? i-1 : i;
	  int xf1 = (i+1 < size.x)? i+1 : i;
	  int xf2 = (i+2 < size.x)? i+2 : i;
	  
	  int yb2 = (j-2 >= 0)? j-2 : j;
	  int yb1 = (j-1 >= 0)? j-1 : j;
	  int yf1 = (j+1 < size.y)? j+1 : j;
	  int yf2 = (j+2 < size.y)? j+2 : j;
	  
	  int zb2 = (k-2 >= 0)? k-2 : k;
	  int zb1 = (k-1 >= 0)? k-1 : k;
	  int zf1 = (k+1 < size.z)? k+1 : k;
	  int zf2 = (k+2 < size.z)? k+2 : k;
	  
	  int comm = j * size.z + k;
	  
	  int4 xn = make_int4(xb2 * size.w + comm, 
						  xb1 * size.w + comm, 
						  xf1 * size.w + comm, 
						  xf2 * size.w + comm); 
						 

	  comm = i * size.w + k; 
	  int4 yn = make_int4(yb2 * size.z + comm, 
						  yb1 * size.z + comm, 
						  yf1 * size.z + comm, 
						  yf2 * size.z + comm);

	  
	  comm = i * size.w + j * size.z;
	  int4 zn = make_int4(zb2 + comm, 
						  zb1 + comm, 
						  zf1 + comm, 
						  zf2 + comm);

	  	
	  // Let's use 5-point stencil to avoid problems at the boundaries
	  // CUDA does not have vec3 operators like GLSL has except of .xxx, 
	  // Perhaps for performance need to take into account special cases where j || to x, y or z  
	  
	  float3 dmdx = 	make_float3(mstep.x * (mx[xn.x] - 8.0f * mx[xn.y] + 8.0f * mx[xn.z] - mx[xn.w]),
									mstep.y * (mx[yn.x] - 8.0f * mx[yn.y] + 8.0f * mx[yn.z] - mx[yn.w]),
									mstep.z * (mx[zn.x] - 8.0f * mx[zn.y] + 8.0f * mx[zn.z] - mx[zn.w]));
								      
	  float3 dmdy = 	make_float3(mstep.x * (my[xn.x] - 8.0f * my[xn.y] + 8.0f * my[xn.z] - my[xn.w]),
								    mstep.y * (my[yn.x] - 8.0f * my[yn.y] + 8.0f * my[yn.z] - my[yn.w]),
									mstep.z * (my[zn.x] - 8.0f * my[zn.y] + 8.0f * my[zn.z] - my[zn.w]));
										
	  float3 dmdz = 	make_float3(mstep.x * (mz[xn.x] - 8.0f * mz[xn.y] + 8.0f * mz[xn.z] - mz[xn.w]),
									mstep.y * (mz[yn.x] - 8.0f * mz[yn.y] + 8.0f * mz[yn.z] - mz[yn.w]),
								    mstep.z * (mz[zn.x] - 8.0f * mz[zn.y] + 8.0f * mz[zn.z] - mz[zn.w]));  
		
	  // Don't see a point of such overkill, nevertheless:
	  
	  float3 j0 = make_float3(0.0f, 0.0f, 0.0f);
	  
	  j0.x = (jx != NULL)? jx[x0] : 0.0f; 
	  j0.y = (jy != NULL)? jy[x0] : 0.0f;  
	  j0.z = (jz != NULL)? jz[x0] : 0.0f;
	  
	  //-------------------------------------------------//
	  		  
	  
	  float3 dmdj = make_float3(dotf(dmdx, j0),
								dotf(dmdy, j0),
								dotf(dmdz, j0));
							
	  
	  
	  
      float3 dmdjxm = crossf(dmdj, m); // with minus in it
      
      float3 mxdmxm = crossf(m, dmdjxm); // with minus from [dmdj x m]
	  	  	  
	  sttx[x0] = m_sat*((pre.x * mxdmxm.x) + (pre.y * dmdjxm.x));
      stty[x0] = m_sat*((pre.x * mxdmxm.y) + (pre.y * dmdjxm.y));
      sttz[x0] = m_sat*((pre.x * mxdmxm.z) + (pre.y * dmdjxm.z));   
    } 
  }

  #define BLOCKSIZE 16
  
__export__  void zhangli_async(float** sttx, float** stty, float** sttz, 
			 float** mx, float** my, float** mz, 
			 float** jx, float** jy, float** jz,
			 float** msat,
			 const float pred, const float pret,
			 const int sx, const int sy, const int sz,
			 const float csx, const float csy, const float csz,
			 int NPart,
			 hipStream_t* stream)
  {

	// 3D :)
	
	dim3 gridSize(divUp(sy, BLOCKSIZE), divUp(sz, BLOCKSIZE));
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
		
	// FUCKING THREADS PER BLOCK LIMITATION
	check3dconf(gridSize, blockSize);
		
	float i12csx = 1.0f / (12.0f * csx);
	float i12csy = 1.0f / (12.0f * csy);
	float i12csz = 1.0f / (12.0f * csz);
	
	int syz = sy * sz;
		
	float3 mstep = make_float3(i12csx, i12csy, i12csz);	
	int4 size = make_int4(sx, sy, sz, syz);
	float2 pre = make_float2(pred, pret);
	
    int nDev = nDevice();
	
	/*hipEvent_t start,stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);*/
	
    for (int dev = 0; dev < nDev; dev++) {
      gpu_safe(hipSetDevice(deviceId(dev)));

		for (int i = 0; i < sx; i++) {
			zhangli_deltaMKern<<<gridSize, blockSize, 0, hipStream_t(stream[dev])>>> (sttx[dev], stty[dev], sttz[dev],  
												   mx[dev], my[dev], mz[dev],											   
												   jx[dev], jy[dev], jz[dev], 
												   msat[dev],
												   pre,
												   size,
												   mstep,
												   i);
		}
    } // end dev < nDev loop
										  
	/*hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("Zhang-Li kernel requires: %f ms\n",time);*/
	
  }
  
  // ========================================

#ifdef __cplusplus
}
#endif
