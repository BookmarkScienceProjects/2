#include "hip/hip_runtime.h"
#include "exchange6.h"

#include "multigpu.h"
#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif
// full 3D blocks
__global__ void exchange6Kern(float* __restrict__ hx, float* __restrict__  hy, float* __restrict__  hz,
                              float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
                              float* __restrict__  mSat_map, float* __restrict__  Aex_map,
                              const float pre,
                              const int N0, const int N1, const int N2,
                              const int wrap0, const int wrap1, const int wrap2,
                              const float cellx_2, const float celly_2, const float cellz_2)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < N0 && j < N1 && k < N2)
    {

        int I = i * N1 * N2 + j * N2 + k;

        float mSat0 = getMaskUnity(mSat_map, I);
        float Aex0 = getMaskUnity(Aex_map, I);
        float lex0Mul = fdivZero(Aex0, mSat0);
        float lexMul, lex1Mul, lex2Mul;

        float mx0 = mx[I]; // mag component of central cell
        float mx1, mx2;

        float my0 = my[I]; // mag component of central cell
        float my1, my2;

        float mz0 = mz[I]; // mag component of central cell
        float mz1, mz2;

        float Hx, Hy, Hz;

        // neighbors in X direction
        int idx = i - 1;
        idx = (idx < 0 && wrap0) ? N0 + idx : idx;
        lexMul = (idx < 0) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex1Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx1 = (idx < 0) ? mx0 : mx[idx * N1 * N2 + j * N2 + k];
        my1 = (idx < 0) ? my0 : my[idx * N1 * N2 + j * N2 + k];
        mz1 = (idx < 0) ? mz0 : mz[idx * N1 * N2 + j * N2 + k];

        idx = i + 1;
        idx = (idx == N0 && wrap0) ? idx - N0 : idx;
        lexMul = (idx == N0) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex2Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx2 = (idx == N0) ? mx0 : mx[idx * N1 * N2 + j * N2 + k];
        my2 = (idx == N0) ? my0 : my[idx * N1 * N2 + j * N2 + k];
        mz2 = (idx == N0) ? mz0 : mz[idx * N1 * N2 + j * N2 + k];

        Hx = pre * cellx_2 * (lex1Mul * (mx1 - mx0) + lex2Mul * (mx2 - mx0));
        Hy = pre * cellx_2 * (lex1Mul * (my1 - my0) + lex2Mul * (my2 - my0));
        Hz = pre * cellx_2 * (lex1Mul * (mz1 - mz0) + lex2Mul * (mz2 - mz0));

        // neighbors in Z direction
        idx = k - 1;
        idx = (idx < 0 && wrap2) ? N2 + idx : idx;
        lexMul = (idx < 0) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex1Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx1 = (idx < 0) ? mx0 : mx[i * N1 * N2 + j * N2 + idx];
        my1 = (idx < 0) ? my0 : my[i * N1 * N2 + j * N2 + idx];
        mz1 = (idx < 0) ? mz0 : mz[i * N1 * N2 + j * N2 + idx];

        idx = k + 1;
        idx = (idx == N2 && wrap2) ? idx - N2 : idx;
        lexMul = (idx == N2) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex2Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx2 = (idx == N2) ? mx0 : mx[i * N1 * N2 + j * N2 + idx];
        my2 = (idx == N2) ? my0 : my[i * N1 * N2 + j * N2 + idx];
        mz2 = (idx == N2) ? mz0 : mz[i * N1 * N2 + j * N2 + idx];

        Hx += pre * cellz_2 * (lex1Mul * (mx1 - mx0) + lex2Mul * (mx2 - mx0));
        Hy += pre * cellz_2 * (lex1Mul * (my1 - my0) + lex2Mul * (my2 - my0));
        Hz += pre * cellz_2 * (lex1Mul * (mz1 - mz0) + lex2Mul * (mz2 - mz0));

        // neighbors in Y direction
        idx = j - 1;
        idx = (idx < 0 && wrap1) ? N1 + idx : idx;
        lexMul = (idx < 0) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex1Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx1 = (idx < 0) ? mx0 : mx[i * N1 * N2 + idx * N2 + k];
        my1 = (idx < 0) ? my0 : my[i * N1 * N2 + idx * N2 + k];
        mz1 = (idx < 0) ? mz0 : mz[i * N1 * N2 + idx * N2 + k];

        idx = j + 1;
        idx = (idx == N1 && wrap1) ? idx - N1 : idx;
        lexMul = (idx == N1) ? 0.0f : fdivZero(getMaskUnity(Aex_map, idx), getMaskUnity(mSat_map, idx));
        lex2Mul = 2.0f * fdivZero((lex0Mul * lexMul), (lex0Mul + lexMul));

        mx2 = (idx == N1) ? mx0 : mx[i * N1 * N2 + idx * N2 + k];
        my2 = (idx == N1) ? my0 : my[i * N1 * N2 + idx * N2 + k];
        mz2 = (idx == N1) ? mz0 : mz[i * N1 * N2 + idx * N2 + k];

        Hx += pre * celly_2 * (lex1Mul * (mx1 - mx0) + lex2Mul * (mx2 - mx0));
        Hy += pre * celly_2 * (lex1Mul * (my1 - my0) + lex2Mul * (my2 - my0));
        Hz += pre * celly_2 * (lex1Mul * (mz1 - mz0) + lex2Mul * (mz2 - mz0));

        // Write back to global memory
        hx[I] = Hx;
        hy[I] = Hy;
        hz[I] = Hz;

    }

}


__export__ void exchange6Async(float** hx, float** hy, float** hz, float** mx, float** my, float** mz, float** msat, float** aex, float Aex2_mu0MsatMul, int N0, int N1Part, int N2, int periodic0, int periodic1, int periodic2, float cellSizeX, float cellSizeY, float cellSizeZ, hipStream_t* streams)
{

    dim3 gridsize, blocksize;

    make3dconf(N0, N1Part, N2, &gridsize, &blocksize);

    float cellx_2 = (float)(1.0 / ((double)cellSizeX * (double)cellSizeX));
    float celly_2 = (float)(1.0 / ((double)cellSizeY * (double)cellSizeY));
    float cellz_2 = (float)(1.0 / ((double)cellSizeZ * (double)cellSizeZ));

    int nDev = nDevice();

    for (int dev = 0; dev < nDev; dev++)
    {
        gpu_safe(hipSetDevice(deviceId(dev)));
        exchange6Kern <<< gridsize, blocksize, 0, hipStream_t(streams[dev])>>>(hx[dev], hy[dev], hz[dev], mx[dev], my[dev], mz[dev], msat[dev], aex[dev], Aex2_mu0MsatMul, N0, N1Part, N2, periodic0, periodic1, periodic2, cellx_2, celly_2, cellz_2);
    }
}


#ifdef __cplusplus
}
#endif

